#include "hip/hip_runtime.h"
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// Make sure we handle target overloads correctly.
// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu \
// RUN:     -fcuda-target-overloads -emit-llvm -o - %s \
// RUN:     | FileCheck -check-prefix=CHECK-BOTH -check-prefix=CHECK-HOST %s
// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fcuda-is-device \
// RUN:     -fcuda-target-overloads -emit-llvm -o - %s \
// RUN:     | FileCheck -check-prefix=CHECK-BOTH -check-prefix=CHECK-DEVICE %s

// Check target overloads handling with disabled call target checks.
// RUN: %clang_cc1 -DNOCHECKS -triple x86_64-unknown-linux-gnu -emit-llvm \
// RUN:    -fcuda-disable-target-call-checks -fcuda-target-overloads -o - %s \
// RUN:     | FileCheck -check-prefix=CHECK-BOTH -check-prefix=CHECK-HOST \
// RUN:    -check-prefix=CHECK-BOTH-NC -check-prefix=CHECK-HOST-NC %s
// RUN: %clang_cc1 -DNOCHECKS -triple nvptx64-nvidia-cuda -emit-llvm \
// RUN:    -fcuda-disable-target-call-checks -fcuda-target-overloads \
// RUN:    -fcuda-is-device -o - %s \
// RUN:     | FileCheck -check-prefix=CHECK-BOTH -check-prefix=CHECK-DEVICE \
// RUN:    -check-prefix=CHECK-BOTH-NC -check-prefix=CHECK-DEVICE-NC %s

#include "Inputs/hip/hip_runtime.h"

typedef int (*fp_t)(void);
typedef void (*gp_t)(void);

// CHECK-HOST: @hp = global i32 ()* @_Z1hv
// CHECK-HOST: @chp = global i32 ()* @ch
// CHECK-HOST: @dhp = global i32 ()* @_Z2dhv
// CHECK-HOST: @cdhp = global i32 ()* @cdh
// CHECK-HOST: @gp = global void ()* @_Z1gv

// CHECK-BOTH-LABEL: define i32 @_Z2dhv()
__device__ int dh(void) { return 1; }
// CHECK-DEVICE: ret i32 1
__host__ int dh(void) { return 2; }
// CHECK-HOST:   ret i32 2

// CHECK-BOTH-LABEL: define i32 @_Z2hdv()
__host__ __device__ int hd(void) { return 3; }
// CHECK-BOTH:   ret i32 3

// CHECK-DEVICE-LABEL: define i32 @_Z1dv()
__device__ int d(void) { return 8; }
// CHECK-DEVICE:   ret i32 8

// CHECK-HOST-LABEL: define i32 @_Z1hv()
__host__ int h(void) { return 9; }
// CHECK-HOST:   ret i32 9

// CHECK-BOTH-LABEL: define void @_Z1gv()
__global__ void g(void) {}
// CHECK-BOTH:   ret void

// mangled names of extern "C" __host__ __device__ functions clash
// with those of their __host__/__device__ counterparts, so
// overloading of extern "C" functions can only happen for __host__
// and __device__ functions -- we never codegen them in the same
// compilation and therefore mangled name conflict is not a problem.

// CHECK-BOTH-LABEL: define i32 @cdh()
extern "C" __device__ int cdh(void) {return 10;}
// CHECK-DEVICE:   ret i32 10
extern "C" __host__ int cdh(void) {return 11;}
// CHECK-HOST:     ret i32 11

// CHECK-DEVICE-LABEL: define i32 @cd()
extern "C" __device__ int cd(void) {return 12;}
// CHECK-DEVICE:   ret i32 12

// CHECK-HOST-LABEL: define i32 @ch()
extern "C" __host__ int ch(void) {return 13;}
// CHECK-HOST:     ret i32 13

// CHECK-BOTH-LABEL: define i32 @chd()
extern "C" __host__ __device__ int chd(void) {return 14;}
// CHECK-BOTH:     ret i32 14

// CHECK-HOST-LABEL: define void @_Z5hostfv()
__host__ void hostf(void) {
#if defined (NOCHECKS)
  fp_t dp = d;   // CHECK-HOST-NC: store {{.*}} @_Z1dv, {{.*}} %dp,
  fp_t cdp = cd; // CHECK-HOST-NC: store {{.*}} @cd, {{.*}} %cdp,
#endif
  fp_t hp = h; // CHECK-HOST: store {{.*}} @_Z1hv, {{.*}} %hp,
  fp_t chp = ch; // CHECK-HOST: store {{.*}} @ch, {{.*}} %chp,
  fp_t dhp = dh; // CHECK-HOST: store {{.*}} @_Z2dhv, {{.*}} %dhp,
  fp_t cdhp = cdh; // CHECK-HOST: store {{.*}} @cdh, {{.*}} %cdhp,
  fp_t hdp = hd; // CHECK-HOST: store {{.*}} @_Z2hdv, {{.*}} %hdp,
  fp_t chdp = chd; // CHECK-HOST: store {{.*}} @chd, {{.*}} %chdp,
  gp_t gp = g; // CHECK-HOST: store {{.*}} @_Z1gv, {{.*}} %gp,

#if defined (NOCHECKS)
  d();     // CHECK-HOST-NC: call i32 @_Z1dv()
  cd();    // CHECK-HOST-NC: call i32 @cd()
#endif
  h();     // CHECK-HOST: call i32 @_Z1hv()
  ch();    // CHECK-HOST: call i32 @ch()
  dh();    // CHECK-HOST: call i32 @_Z2dhv()
  cdh();   // CHECK-HOST: call i32 @cdh()
  g<<<0,0>>>();  // CHECK-HOST: call void @_Z1gv()
}

// CHECK-DEVICE-LABEL: define void @_Z7devicefv()
__device__ void devicef(void) {
  fp_t dp = d;   // CHECK-DEVICE: store {{.*}} @_Z1dv, {{.*}} %dp,
  fp_t cdp = cd; // CHECK-DEVICE: store {{.*}} @cd, {{.*}} %cdp,
#if defined (NOCHECKS)
  fp_t hp = h; // CHECK-DEVICE-NC: store {{.*}} @_Z1hv, {{.*}} %hp,
  fp_t chp = ch; // CHECK-DEVICE-NC: store {{.*}} @ch, {{.*}} %chp,
#endif
  fp_t dhp = dh; // CHECK-DEVICE: store {{.*}} @_Z2dhv, {{.*}} %dhp,
  fp_t cdhp = cdh; // CHECK-DEVICE: store {{.*}} @cdh, {{.*}} %cdhp,
  fp_t hdp = hd; // CHECK-DEVICE: store {{.*}} @_Z2hdv, {{.*}} %hdp,
  fp_t chdp = chd; // CHECK-DEVICE: store {{.*}} @chd, {{.*}} %chdp,

  d();     // CHECK-DEVICE: call i32 @_Z1dv()
  cd();    // CHECK-DEVICE: call i32 @cd()
#if defined (NOCHECKS)
  h();     // CHECK-DEVICE-NC: call i32 @_Z1hv()
  ch();    // CHECK-DEVICE-NC: call i32 @ch()
#endif
  dh();    // CHECK-DEVICE: call i32 @_Z2dhv()
  cdh();   // CHECK-DEVICE: call i32 @cdh()
}

// CHECK-BOTH-LABEL: define void @_Z11hostdevicefv()
__host__ __device__ void hostdevicef(void) {
#if defined (NOCHECKS)
  fp_t dp = d;   // CHECK-BOTH-NC: store {{.*}} @_Z1dv, {{.*}} %dp,
  fp_t cdp = cd; // CHECK-BOTH-NC: store {{.*}} @cd, {{.*}} %cdp,
  fp_t hp = h; // CHECK-BOTH-NC: store {{.*}} @_Z1hv, {{.*}} %hp,
  fp_t chp = ch; // CHECK-BOTH-NC: store {{.*}} @ch, {{.*}} %chp,
#endif
  fp_t dhp = dh; // CHECK-BOTH: store {{.*}} @_Z2dhv, {{.*}} %dhp,
  fp_t cdhp = cdh; // CHECK-BOTH: store {{.*}} @cdh, {{.*}} %cdhp,
  fp_t hdp = hd; // CHECK-BOTH: store {{.*}} @_Z2hdv, {{.*}} %hdp,
  fp_t chdp = chd; // CHECK-BOTH: store {{.*}} @chd, {{.*}} %chdp,
#if defined (NOCHECKS) && !defined(__CUDA_ARCH__)
  gp_t gp = g; // CHECK-HOST-NC: store {{.*}} @_Z1gv, {{.*}} %gp,
#endif

#if defined (NOCHECKS)
  d();     // CHECK-BOTH-NC: call i32 @_Z1dv()
  cd();    // CHECK-BOTH-NC: call i32 @cd()
  h();     // CHECK-BOTH-NC: call i32 @_Z1hv()
  ch();    // CHECK-BOTH-NC: call i32 @ch()
#endif
  dh();    // CHECK-BOTH: call i32 @_Z2dhv()
  cdh();   // CHECK-BOTH: call i32 @cdh()
#if defined (NOCHECKS) && !defined(__CUDA_ARCH__)
  g<<<0,0>>>();  // CHECK-HOST-NC: call void @_Z1gv()
#endif
}

// Test for address of overloaded function resolution in the global context.
fp_t hp = h;
fp_t chp = ch;
fp_t dhp = dh;
fp_t cdhp = cdh;
gp_t gp = g;

int x;
// Check constructors/destructors for D/H functions
struct s_cd_dh {
  __host__ s_cd_dh() { x = 11; }
  __device__ s_cd_dh() { x = 12; }
  __host__ ~s_cd_dh() { x = 21; }
  __device__ ~s_cd_dh() { x = 22; }
};

struct s_cd_hd {
  __host__ __device__ s_cd_hd() { x = 31; }
  __host__ __device__ ~s_cd_hd() { x = 32; }
};

// CHECK-BOTH: define void @_Z7wrapperv
#if defined(__CUDA_ARCH__)
__device__
#else
__host__
#endif
void wrapper() {
  s_cd_dh scddh;
  // CHECK-BOTH: call void @_ZN7s_cd_dhC1Ev(
  s_cd_hd scdhd;
  // CHECK-BOTH: call void @_ZN7s_cd_hdC1Ev

  // CHECK-BOTH: call void @_ZN7s_cd_hdD1Ev(
  // CHECK-BOTH: call void @_ZN7s_cd_dhD1Ev(
}
// CHECK-BOTH: ret void

// Now it's time to check what's been generated for the methods we used.

// CHECK-BOTH: define linkonce_odr void @_ZN7s_cd_dhC2Ev(
// CHECK-HOST:   store i32 11,
// CHECK-DEVICE: store i32 12,
// CHECK-BOTH: ret void

// CHECK-BOTH: define linkonce_odr void @_ZN7s_cd_hdC2Ev(
// CHECK-BOTH:   store i32 31,
// CHECK-BOTH: ret void

// CHECK-BOTH: define linkonce_odr void @_ZN7s_cd_hdD2Ev(
// CHECK-BOTH: store i32 32,
// CHECK-BOTH: ret void

// CHECK-BOTH: define linkonce_odr void @_ZN7s_cd_dhD2Ev(
// CHECK-HOST:   store i32 21,
// CHECK-DEVICE: store i32 22,
// CHECK-BOTH: ret void

