#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -emit-llvm %s -o - -fcuda-is-device -triple nvptx-unknown-unknown | FileCheck %s

#include "../SemaCUDA/hip/hip_runtime.h"

// CHECK: @i = addrspace(1) global
__device__ int i;

// CHECK: @j = addrspace(4) global
__constant__ int j;

// CHECK: @k = addrspace(3) global
__shared__ int k;

__device__ void foo() {
  // CHECK: load i32* addrspacecast (i32 addrspace(1)* @i to i32*)
  i++;

  // CHECK: load i32* addrspacecast (i32 addrspace(4)* @j to i32*)
  j++;

  // CHECK: load i32* addrspacecast (i32 addrspace(3)* @k to i32*)
  k++;

  static int li;
  // CHECK: load i32 addrspace(1)* @_ZZ3foovE2li
  li++;

  __constant__ int lj;
  // CHECK: load i32 addrspace(4)* @_ZZ3foovE2lj
  lj++;

  __shared__ int lk;
  // CHECK: load i32 addrspace(3)* @_ZZ3foovE2lk
  lk++;
}

