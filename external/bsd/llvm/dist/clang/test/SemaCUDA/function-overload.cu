#include "hip/hip_runtime.h"
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// Make sure we handle target overloads correctly.
// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu \
// RUN:    -fsyntax-only -fcuda-target-overloads -verify %s
// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda \
// RUN:    -fsyntax-only -fcuda-target-overloads -fcuda-is-device -verify %s

// Check target overloads handling with disabled call target checks.
// RUN: %clang_cc1 -DNOCHECKS -triple x86_64-unknown-linux-gnu -fsyntax-only \
// RUN:    -fcuda-disable-target-call-checks -fcuda-target-overloads -verify %s
// RUN: %clang_cc1 -DNOCHECKS -triple nvptx64-nvidia-cuda -fsyntax-only \
// RUN:    -fcuda-disable-target-call-checks -fcuda-target-overloads \
// RUN:    -fcuda-is-device -verify %s

#include "Inputs/hip/hip_runtime.h"

typedef int (*fp_t)(void);
typedef void (*gp_t)(void);

// Host and unattributed functions can't be overloaded
__host__ int hh(void) { return 1; } // expected-note {{previous definition is here}}
int hh(void) { return 1; } // expected-error {{redefinition of 'hh'}}

// H/D overloading is OK
__host__ int dh(void) { return 2; }
__device__ int dh(void) { return 2; }

// H/HD and D/HD are not allowed
__host__ __device__ int hdh(void) { return 5; } // expected-note {{previous definition is here}}
__host__ int hdh(void) { return 4; } // expected-error {{redefinition of 'hdh'}}

__host__ int hhd(void) { return 4; } // expected-note {{previous definition is here}}
__host__ __device__ int hhd(void) { return 5; } // expected-error {{redefinition of 'hhd'}}
// expected-warning@-1 {{attribute declaration must precede definition}}
// expected-note@-3 {{previous definition is here}}

__host__ __device__ int hdd(void) { return 7; } // expected-note {{previous definition is here}}
__device__ int hdd(void) { return 6; } // expected-error {{redefinition of 'hdd'}}

__device__ int dhd(void) { return 6; } // expected-note {{previous definition is here}}
__host__ __device__ int dhd(void) { return 7; } // expected-error {{redefinition of 'dhd'}}
// expected-warning@-1 {{attribute declaration must precede definition}}
// expected-note@-3 {{previous definition is here}}

// Same tests for extern "C" functions
extern "C" __host__ int chh(void) {return 11;} // expected-note {{previous definition is here}}
extern "C" int chh(void) {return 11;} // expected-error {{redefinition of 'chh'}}

// H/D overloading is OK
extern "C" __device__ int cdh(void) {return 10;}
extern "C" __host__ int cdh(void) {return 11;}

// H/HD and D/HD overloading is not allowed.
extern "C" __host__ __device__ int chhd1(void) {return 12;} // expected-note {{previous definition is here}}
extern "C" __host__ int chhd1(void) {return 13;} // expected-error {{redefinition of 'chhd1'}}

extern "C" __host__ int chhd2(void) {return 13;} // expected-note {{previous definition is here}}
extern "C" __host__ __device__ int chhd2(void) {return 12;} // expected-error {{redefinition of 'chhd2'}}
// expected-warning@-1 {{attribute declaration must precede definition}}
// expected-note@-3 {{previous definition is here}}

// Helper functions to verify calling restrictions.
__device__ int d(void) { return 8; }
__host__ int h(void) { return 9; }
__global__ void g(void) {}
extern "C" __device__ int cd(void) {return 10;}
extern "C" __host__ int ch(void) {return 11;}

__host__ void hostf(void) {
  fp_t dp = d;
  fp_t cdp = cd;
#if !defined(NOCHECKS)
  // expected-error@-3 {{reference to __device__ function 'd' in __host__ function}}
  // expected-note@65 {{'d' declared here}}
  // expected-error@-4 {{reference to __device__ function 'cd' in __host__ function}}
  // expected-note@68 {{'cd' declared here}}
#endif
  fp_t hp = h;
  fp_t chp = ch;
  fp_t dhp = dh;
  fp_t cdhp = cdh;
  gp_t gp = g;

  d();
  cd();
#if !defined(NOCHECKS)
  // expected-error@-3 {{no matching function for call to 'd'}}
  // expected-note@65 {{candidate function not viable: call to __device__ function from __host__ function}}
  // expected-error@-4 {{no matching function for call to 'cd'}}
  // expected-note@68 {{candidate function not viable: call to __device__ function from __host__ function}}
#endif
  h();
  ch();
  dh();
  cdh();
  g(); // expected-error {{call to global function g not configured}}
  g<<<0,0>>>();
}


__device__ void devicef(void) {
  fp_t dp = d;
  fp_t cdp = cd;
  fp_t hp = h;
  fp_t chp = ch;
#if !defined(NOCHECKS)
  // expected-error@-3 {{reference to __host__ function 'h' in __device__ function}}
  // expected-note@66 {{'h' declared here}}
  // expected-error@-4 {{reference to __host__ function 'ch' in __device__ function}}
  // expected-note@69 {{'ch' declared here}}
#endif
  fp_t dhp = dh;
  fp_t cdhp = cdh;
  gp_t gp = g; // expected-error {{reference to __global__ function 'g' in __device__ function}}
               // expected-note@67 {{'g' declared here}}

  d();
  cd();
  h();
  ch();
#if !defined(NOCHECKS)
  // expected-error@-3 {{no matching function for call to 'h'}}
  // expected-note@66 {{candidate function not viable: call to __host__ function from __device__ function}}
  // expected-error@-4 {{no matching function for call to 'ch'}}
  // expected-note@69 {{candidate function not viable: call to __host__ function from __device__ function}}
#endif
  dh();
  cdh();
  g(); // expected-error {{no matching function for call to 'g'}}
  // expected-note@67 {{candidate function not viable: call to __global__ function from __device__ function}}
  g<<<0,0>>>(); // expected-error {{reference to __global__ function 'g' in __device__ function}}
  // expected-note@67 {{'g' declared here}}
}

__global__ void globalf(void) {
  fp_t dp = d;
  fp_t cdp = cd;
  fp_t hp = h;
  fp_t chp = ch;
#if !defined(NOCHECKS)
  // expected-error@-3 {{reference to __host__ function 'h' in __global__ function}}
  // expected-note@66 {{'h' declared here}}
  // expected-error@-4 {{reference to __host__ function 'ch' in __global__ function}}
  // expected-note@69 {{'ch' declared here}}
#endif
  fp_t dhp = dh;
  fp_t cdhp = cdh;
  gp_t gp = g; // expected-error {{reference to __global__ function 'g' in __global__ function}}
               // expected-note@67 {{'g' declared here}}

  d();
  cd();
  h();
  ch();
#if !defined(NOCHECKS)
  // expected-error@-3 {{no matching function for call to 'h'}}
  // expected-note@66 {{candidate function not viable: call to __host__ function from __global__ function}}
  // expected-error@-4 {{no matching function for call to 'ch'}}
  // expected-note@69 {{candidate function not viable: call to __host__ function from __global__ function}}
#endif
  dh();
  cdh();
  g(); // expected-error {{no matching function for call to 'g'}}
  // expected-note@67 {{candidate function not viable: call to __global__ function from __global__ function}}
  g<<<0,0>>>(); // expected-error {{reference to __global__ function 'g' in __global__ function}}
  // expected-note@67 {{'g' declared here}}
}

__host__ __device__ void hostdevicef(void) {
  fp_t dp = d;
  fp_t cdp = cd;
  fp_t hp = h;
  fp_t chp = ch;
#if !defined(NOCHECKS)
#if !defined(__CUDA_ARCH__)
  // expected-error@-6 {{reference to __device__ function 'd' in __host__ __device__ function}}
  // expected-note@65 {{'d' declared here}}
  // expected-error@-7 {{reference to __device__ function 'cd' in __host__ __device__ function}}
  // expected-note@68 {{'cd' declared here}}
#else
  // expected-error@-9 {{reference to __host__ function 'h' in __host__ __device__ function}}
  // expected-note@66 {{'h' declared here}}
  // expected-error@-10 {{reference to __host__ function 'ch' in __host__ __device__ function}}
  // expected-note@69 {{'ch' declared here}}
#endif
#endif
  fp_t dhp = dh;
  fp_t cdhp = cdh;
  gp_t gp = g;
#if defined(__CUDA_ARCH__)
  // expected-error@-2 {{reference to __global__ function 'g' in __host__ __device__ function}}
  // expected-note@67 {{'g' declared here}}
#endif

  d();
  cd();
  h();
  ch();
#if !defined(NOCHECKS)
#if !defined(__CUDA_ARCH__)
  // expected-error@-6 {{no matching function for call to 'd'}}
  // expected-note@65 {{candidate function not viable: call to __device__ function from __host__ __device__ function}}
  // expected-error@-7 {{no matching function for call to 'cd'}}
  // expected-note@68 {{candidate function not viable: call to __device__ function from __host__ __device__ function}}
#else
  // expected-error@-9 {{no matching function for call to 'h'}}
  // expected-note@66 {{candidate function not viable: call to __host__ function from __host__ __device__ function}}
  // expected-error@-10 {{no matching function for call to 'ch'}}
  // expected-note@69 {{candidate function not viable: call to __host__ function from __host__ __device__ function}}
#endif
#endif

  dh();
  cdh();
  g();
  g<<<0,0>>>();
#if !defined(__CUDA_ARCH__)
  // expected-error@-3 {{call to global function g not configured}}
#else
  // expected-error@-5 {{no matching function for call to 'g'}}
  // expected-note@67 {{candidate function not viable: call to __global__ function from __host__ __device__ function}}
  // expected-error@-6 {{reference to __global__ function 'g' in __host__ __device__ function}}
  // expected-note@67 {{'g' declared here}}
#endif  // __CUDA_ARCH__
}

// Test for address of overloaded function resolution in the global context.
fp_t hp = h;
fp_t chp = ch;
fp_t dhp = dh;
fp_t cdhp = cdh;
gp_t gp = g;


// Test overloading of destructors
// Can't mix H and unattributed destructors
struct d_h {
  ~d_h() {} // expected-note {{previous declaration is here}}
  __host__ ~d_h() {} // expected-error {{destructor cannot be redeclared}}
};

// H/D overloading is OK
struct d_dh {
  __device__ ~d_dh() {}
  __host__ ~d_dh() {}
};

// HD is OK
struct d_hd {
  __host__ __device__ ~d_hd() {}
};

// Mixing H/D and HD is not allowed.
struct d_dhhd {
  __device__ ~d_dhhd() {}
  __host__ ~d_dhhd() {} // expected-note {{previous declaration is here}}
  __host__ __device__ ~d_dhhd() {} // expected-error {{destructor cannot be redeclared}}
};

struct d_hhd {
  __host__ ~d_hhd() {} // expected-note {{previous declaration is here}}
  __host__ __device__ ~d_hhd() {} // expected-error {{destructor cannot be redeclared}}
};

struct d_hdh {
  __host__ __device__ ~d_hdh() {} // expected-note {{previous declaration is here}}
  __host__ ~d_hdh() {} // expected-error {{destructor cannot be redeclared}}
};

struct d_dhd {
  __device__ ~d_dhd() {} // expected-note {{previous declaration is here}}
  __host__ __device__ ~d_dhd() {} // expected-error {{destructor cannot be redeclared}}
};

struct d_hdd {
  __host__ __device__ ~d_hdd() {} // expected-note {{previous declaration is here}}
  __device__ ~d_hdd() {} // expected-error {{destructor cannot be redeclared}}
};

// Test overloading of member functions
struct m_h {
  void operator delete(void *ptr); // expected-note {{previous declaration is here}}
  __host__ void operator delete(void *ptr); // expected-error {{class member cannot be redeclared}}
};

// D/H overloading is OK
struct m_dh {
  __device__ void operator delete(void *ptr);
  __host__ void operator delete(void *ptr);
};

// HD by itself is OK
struct m_hd {
  __device__ __host__ void operator delete(void *ptr);
};

struct m_hhd {
  __host__ void operator delete(void *ptr) {} // expected-note {{previous declaration is here}}
  __host__ __device__ void operator delete(void *ptr) {} // expected-error {{class member cannot be redeclared}}
};

struct m_hdh {
  __host__ __device__ void operator delete(void *ptr) {} // expected-note {{previous declaration is here}}
  __host__ void operator delete(void *ptr) {} // expected-error {{class member cannot be redeclared}}
};

struct m_dhd {
  __device__ void operator delete(void *ptr) {} // expected-note {{previous declaration is here}}
  __host__ __device__ void operator delete(void *ptr) {} // expected-error {{class member cannot be redeclared}}
};

struct m_hdd {
  __host__ __device__ void operator delete(void *ptr) {} // expected-note {{previous declaration is here}}
  __device__ void operator delete(void *ptr) {} // expected-error {{class member cannot be redeclared}}
};
