#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -fsyntax-only -verify %s

#include "Inputs/hip/hip_runtime.h"

__host__ void h1h(void);
__device__ void h1d(void); // expected-note {{candidate function not viable: call to __device__ function from __host__ function}}
__host__ __device__ void h1hd(void);
__global__ void h1g(void);

struct h1ds { // expected-note {{requires 1 argument}}
  __device__ h1ds(); // expected-note {{candidate constructor not viable: call to __device__ function from __host__ function}}
};

__host__ void h1(void) {
  h1h();
  h1d(); // expected-error {{no matching function}}
  h1hd();
  h1g<<<1, 1>>>();
  h1ds x; // expected-error {{no matching constructor}}
}

__host__ void d1h(void); // expected-note {{candidate function not viable: call to __host__ function from __device__ function}}
__device__ void d1d(void);
__host__ __device__ void d1hd(void);
__global__ void d1g(void); // expected-note {{'d1g' declared here}}

__device__ void d1(void) {
  d1h(); // expected-error {{no matching function}}
  d1d();
  d1hd();
  d1g<<<1, 1>>>(); // expected-error {{reference to __global__ function 'd1g' in __device__ function}}
}

__host__ void hd1h(void); // expected-note {{candidate function not viable: call to __host__ function from __host__ __device__ function}}
__device__ void hd1d(void); // expected-note {{candidate function not viable: call to __device__ function from __host__ __device__ function}}
__host__ __device__ void hd1hd(void);
__global__ void hd1g(void); // expected-note {{'hd1g' declared here}}

__host__ __device__ void hd1(void) {
  hd1h(); // expected-error {{no matching function}}
  hd1d(); // expected-error {{no matching function}}
  hd1hd();
  hd1g<<<1, 1>>>(); // expected-error {{reference to __global__ function 'hd1g' in __host__ __device__ function}}
}
