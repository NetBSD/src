
#include <hip/hip_runtime.h>
// REQUIRES: clang-driver
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target
//
// # Check that we properly detect CUDA installation.
// RUN: %clang -v --target=i386-unknown-linux \
// RUN:   --sysroot=%S/no-cuda-there 2>&1 | FileCheck %s -check-prefix NOCUDA
// RUN: %clang -v --target=i386-unknown-linux \
// RUN:   --sysroot=%S/Inputs/CUDA 2>&1 | FileCheck %s
// RUN: %clang -v --target=i386-unknown-linux \
// RUN:   --cuda-path=%S/Inputs/CUDA/usr/local/cuda 2>&1 | FileCheck %s

// Make sure we map libdevice bitcode files to proper GPUs.
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_21 \
// RUN:   --cuda-path=%S/Inputs/CUDA/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON \
// RUN:     -check-prefix LIBDEVICE -check-prefix LIBDEVICE21
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_35 \
// RUN:   --cuda-path=%S/Inputs/CUDA/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON -check-prefix CUDAINC \
// RUN:     -check-prefix LIBDEVICE -check-prefix LIBDEVICE35
// Verify that -nocudainc prevents adding include path to CUDA headers.
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_35 \
// RUN:   -nocudainc --cuda-path=%S/Inputs/CUDA/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON -check-prefix NOCUDAINC \
// RUN:     -check-prefix LIBDEVICE -check-prefix LIBDEVICE35
// We should not add any CUDA include paths if there's no valid CUDA installation
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_35 \
// RUN:   --cuda-path=%S/no-cuda-there %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON -check-prefix NOCUDAINC

// Verify that no options related to bitcode linking are passes if
// there's no bitcode file.
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_30 \
// RUN:   --cuda-path=%S/Inputs/CUDA/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON -check-prefix NOLIBDEVICE
// .. or if we explicitly passed -nocudalib
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_35 \
// RUN:   -nocudalib --cuda-path=%S/Inputs/CUDA/usr/local/cuda %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON -check-prefix NOLIBDEVICE
// Verify that we don't add include paths, link with libdevice or
// -include __clang_cuda_runtime_wrapper.h without valid CUDA installation.
// RUN: %clang -### -v --target=i386-unknown-linux --cuda-gpu-arch=sm_35 \
// RUN:   --cuda-path=%S/no-cuda-there %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix COMMON \
// RUN:     -check-prefix NOCUDAINC -check-prefix NOLIBDEVICE

// CHECK: Found CUDA installation: {{.*}}/Inputs/CUDA/usr/local/cuda
// NOCUDA-NOT: Found CUDA installation:

// COMMON: "-triple" "nvptx-nvidia-cuda"
// COMMON-SAME: "-fcuda-is-device"
// LIBDEVICE-SAME: "-mlink-cuda-bitcode"
// NOLIBDEVICE-NOT: "-mlink-cuda-bitcode"
// LIBDEVICE21-SAME: libdevice.compute_20.10.bc
// LIBDEVICE35-SAME: libdevice.compute_35.10.bc
// NOLIBDEVICE-NOT: libdevice.compute_{{.*}}.bc
// LIBDEVICE-SAME: "-target-feature" "+ptx42"
// NOLIBDEVICE-NOT: "-target-feature" "+ptx42"
// CUDAINC-SAME: "-internal-isystem" "{{.*}}/Inputs/CUDA/usr/local/cuda/include"
// NOCUDAINC-NOT: "-internal-isystem" "{{.*}}/cuda/include"
// CUDAINC-SAME: "-include" "__clang_cuda_runtime_wrapper.h"
// NOCUDAINC-NOT: "-include" "__clang_cuda_runtime_wrapper.h"
// COMMON-SAME: "-x" "cuda"
