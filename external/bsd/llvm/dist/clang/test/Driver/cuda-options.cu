
#include <hip/hip_runtime.h>
// Tests CUDA compilation pipeline construction in Driver.
// REQUIRES: clang-driver
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// Simple compilation case. Compile device-side to PTX assembly and make sure
// we use it on the host side.
// RUN: %clang -### -target x86_64-linux-gnu -c %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix HOST -check-prefix INCLUDES-DEVICE \
// RUN:    -check-prefix NOLINK %s

// Typical compilation + link case.
// RUN: %clang -### -target x86_64-linux-gnu %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix HOST -check-prefix INCLUDES-DEVICE \
// RUN:    -check-prefix LINK %s

// Verify that --cuda-host-only disables device-side compilation, but doesn't
// disable host-side compilation/linking.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-host-only %s 2>&1 \
// RUN: | FileCheck -check-prefix NODEVICE -check-prefix HOST \
// RUN:    -check-prefix NOINCLUDES-DEVICE -check-prefix LINK %s

// Same test as above, but with preceeding --cuda-device-only to make sure only
// the last option has an effect.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-device-only --cuda-host-only %s 2>&1 \
// RUN: | FileCheck -check-prefix NODEVICE -check-prefix HOST \
// RUN:    -check-prefix NOINCLUDES-DEVICE -check-prefix LINK %s

// Verify that --cuda-device-only disables host-side compilation and linking.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-device-only %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix NOHOST -check-prefix NOLINK %s

// Same test as above, but with preceeding --cuda-host-only to make sure only
// the last option has an effect.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-host-only --cuda-device-only %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix NOHOST -check-prefix NOLINK %s

// Verify that with -S we compile host and device sides to assembly and
// incorporate device code into the host side.
// RUN: %clang -### -target x86_64-linux-gnu -S -c %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix HOST -check-prefix INCLUDES-DEVICE \
// RUN:    -check-prefix NOLINK %s

// Verify that --cuda-gpu-arch option passes the correct GPU archtecture to
// device compilation.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-gpu-arch=sm_35 -c %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix DEVICE-SM35 -check-prefix HOST \
// RUN:    -check-prefix INCLUDES-DEVICE -check-prefix NOLINK %s

// Verify that there is one device-side compilation per --cuda-gpu-arch args
// and that all results are included on the host side.
// RUN: %clang -### -target x86_64-linux-gnu \
// RUN:   --cuda-gpu-arch=sm_35 --cuda-gpu-arch=sm_30 -c %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix DEVICE2 -check-prefix DEVICE-SM35 \
// RUN:    -check-prefix DEVICE2-SM30 -check-prefix HOST \
// RUN:    -check-prefix HOST-NOSAVE -check-prefix INCLUDES-DEVICE \
// RUN:    -check-prefix INCLUDES-DEVICE2 -check-prefix NOLINK %s

// Verify that device-side results are passed to the correct tool when
// -save-temps is used.
// RUN: %clang -### -target x86_64-linux-gnu -save-temps -c %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-SAVE \
// RUN:    -check-prefix HOST -check-prefix HOST-SAVE -check-prefix NOLINK %s

// Verify that device-side results are passed to the correct tool when
// -fno-integrated-as is used.
// RUN: %clang -### -target x86_64-linux-gnu -fno-integrated-as -c %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix HOST -check-prefix HOST-NOSAVE \
// RUN:    -check-prefix HOST-AS -check-prefix NOLINK %s

// Match device-side preprocessor and compiler phases with -save-temps.
// DEVICE-SAVE: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// DEVICE-SAVE-SAME: "-aux-triple" "x86_64--linux-gnu"
// DEVICE-SAVE-SAME: "-fcuda-is-device"
// DEVICE-SAVE-SAME: "-x" "cuda"

// DEVICE-SAVE: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// DEVICE-SAVE-SAME: "-aux-triple" "x86_64--linux-gnu"
// DEVICE-SAVE-SAME: "-fcuda-is-device"
// DEVICE-SAVE-SAME: "-x" "cuda-cpp-output"

// Match the job that produces PTX assembly.
// DEVICE: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// DEVICE-NOSAVE-SAME: "-aux-triple" "x86_64--linux-gnu"
// DEVICE-SAME: "-fcuda-is-device"
// DEVICE-SM35-SAME: "-target-cpu" "sm_35"
// DEVICE-SAME: "-o" "[[GPUBINARY1:[^"]*]]"
// DEVICE-NOSAVE-SAME: "-x" "cuda"
// DEVICE-SAVE-SAME: "-x" "ir"

// Match another device-side compilation.
// DEVICE2: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// DEVICE2-SAME: "-aux-triple" "x86_64--linux-gnu"
// DEVICE2-SAME: "-fcuda-is-device"
// DEVICE2-SM30-SAME: "-target-cpu" "sm_30"
// DEVICE2-SAME: "-o" "[[GPUBINARY2:[^"]*]]"
// DEVICE2-SAME: "-x" "cuda"

// Match no device-side compilation.
// NODEVICE-NOT: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// NODEVICE-SAME-NOT: "-fcuda-is-device"

// Match host-side preprocessor job with -save-temps.
// HOST-SAVE: "-cc1" "-triple" "x86_64--linux-gnu"
// HOST-SAVE-SAME: "-aux-triple" "nvptx64-nvidia-cuda"
// HOST-SAVE-SAME-NOT: "-fcuda-is-device"
// HOST-SAVE-SAME: "-x" "cuda"

// Match host-side compilation.
// HOST: "-cc1" "-triple" "x86_64--linux-gnu"
// HOST-SAME: "-aux-triple" "nvptx64-nvidia-cuda"
// HOST-SAME-NOT: "-fcuda-is-device"
// HOST-SAME: "-o" "[[HOSTOUTPUT:[^"]*]]"
// HOST-NOSAVE-SAME: "-x" "cuda"
// HOST-SAVE-SAME: "-x" "cuda-cpp-output"
// INCLUDES-DEVICE-SAME: "-fcuda-include-gpubinary" "[[GPUBINARY1]]"
// INCLUDES-DEVICE2-SAME: "-fcuda-include-gpubinary" "[[GPUBINARY2]]"

// Match external assembler that uses compilation output.
// HOST-AS: "-o" "{{.*}}.o" "[[HOSTOUTPUT]]"

// Match no GPU code inclusion.
// NOINCLUDES-DEVICE-NOT: "-fcuda-include-gpubinary"

// Match no host compilation.
// NOHOST-NOT: "-cc1" "-triple"
// NOHOST-SAME-NOT: "-x" "cuda"

// Match linker.
// LINK: "{{.*}}{{ld|link}}{{(.exe)?}}"
// LINK-SAME: "[[HOSTOUTPUT]]"

// Match no linker.
// NOLINK-NOT: "{{.*}}{{ld|link}}{{(.exe)?}}"
